#include "hip/hip_runtime.h"
//
// Cuda Beam solver functions 
//

#include<CudaDeviceFunctions.h>

__global__ void dev_advance(DeviceBeamSolver dev_beamsolver, DeviceUndulator dev_und, DeviceField dev_field, Particle* particles, unsigned int size){

    hipDoubleComplex* rpart = (hipDoubleComplex*)malloc(dev_beamsolver.numel*sizeof(hipDoubleComplex));
    __shared__ Particle local_parts[BLOCK_DIM];

    double awloc;
    double btpar;
    double wx, wy;
    unsigned int nfld;
    hipDoubleComplex cpart; 
    int islice, idx_part = 0;
    unsigned int idx = BLOCK_DIM*blockIdx.x+threadIdx.x;

    dev_move_particles(particles+BLOCK_DIM*blockIdx.x, local_parts);
    __syncthreads();

    if (idx < size) {

        Particle part = local_parts[threadIdx.x];
        awloc = dev_faw2(part.x, part.y, dev_und);
        btpar=1+part.px*part.px+part.py*part.py+dev_und.aw*dev_und.aw*awloc*awloc;	  
        for (unsigned int ifld=0; ifld<dev_beamsolver.numel; ifld++){
            islice = (idx + first_arr[ifld]) % dev_field.xsize;
            nfld = nfld_arr[ifld];
            if(dev_getLLGridpoint(part.x, part.y, wx, wy, idx_part, dev_field, nfld)){
                cpart = cuCxR(tex3DDoubleComplex(nfld, islice, idx_part), wx*wy);
                idx_part++;
                cuCAddAss(cpart, cuCxR(tex3DDoubleComplex(nfld, islice, idx_part),(1-wx)*wy));
                idx_part += ngrid_arr[nfld] - 1;
                cuCAddAss(cpart, cuCxR(tex3DDoubleComplex(nfld, islice, idx_part),wx*(1-wy)));
                idx_part++;
                cuCAddAss(cpart, cuCxR(tex3DDoubleComplex(nfld, islice, idx_part),(1-wx)*(1-wy)));
                cpart = hipConj(cpart);
                awloc *= rtmp_arr[ifld];
                cpart = cuCxR(cpart, awloc);
                rpart[ifld] = cpart;
            } else {
                rpart[ifld] = make_hipDoubleComplex(0, 0);
            }
        }
        dev_rungekutta(part.gamma, part.theta, btpar, rpart, dev_beamsolver); 
        local_parts[threadIdx.x] = part;
    }

    free(rpart);
    __syncthreads(); 
    dev_move_particles(local_parts,particles+BLOCK_DIM*blockIdx.x);
    __syncthreads(); 
}

__device__ hipDoubleComplex cuCxR (hipDoubleComplex x, double y)
{
    hipDoubleComplex ret = x;
    ret.x *= y;
    ret.y *= y;
    return ret;
}

__device__ void cuCAddAss (hipDoubleComplex &x, hipDoubleComplex y) { 
    x.x += y.x;
    x.y += y.y;    
}

__device__ hipDoubleComplex tex3DDoubleComplex(unsigned int i, unsigned int j, unsigned int k) {

    hipDoubleComplex ret;
    int4 v = surf2DLayeredread<int4>(field_surf, j, k, i);
    ret.x = __hiloint2double(v.y, v.x);
    ret.y = __hiloint2double(v.w, v.z);
    return ret;
}


__device__ void dev_move_particles(Particle* src, Particle* dst){

    double* src_array = (double*) src;
    double* dst_array = (double*) dst;

    for (unsigned int i = 0; i < 6; ++i) {
        dst_array[BLOCK_DIM*i+threadIdx.x] = src_array[BLOCK_DIM*i+threadIdx.x];
    }
}

__device__ double dev_faw2(double x, double y, DeviceUndulator& dev_und){  // square of the transverse dependence of the undulator field.
    double dx=x-dev_und.ax;
    double dy=y-dev_und.ay; 
    return (1+dev_und.kx*dx*dx+dev_und.ky*dy*dy+2*(dev_und.gradx*dx+dev_und.grady*dy));
}


__device__ bool dev_getLLGridpoint(double x, double y, double &wx, double &wy, int &idx, DeviceField& dev_field, unsigned int field_idx){

    double gridmax = gridmax_arr[field_idx];
    double dgrid   = dgrid_arr[field_idx];
    int ngrid      = ngrid_arr[field_idx];

    bool ret = (fabs(x) < gridmax) && (fabs(y) < gridmax);
    if (ret) {
        double tempwx = (x+gridmax)/dgrid;
        double tempwy = (y+gridmax)/dgrid;
        tempwx = 1+floor(tempwx)-tempwx; 
        tempwy = 1+floor(tempwy)-tempwy; 

        int ix = static_cast<int> (floor(tempwx));
        int iy = static_cast<int> (floor(tempwy));
        int tempidx = ix+iy*ngrid;

        wx  = tempwx;
        wy  = tempwy;
        idx = tempidx;
    }
    return ret;
}

__device__ void dev_run_ODE(double gamma, double theta, double& k2pp, double& k2gg, double btpar, hipDoubleComplex* rpart, DeviceBeamSolver dev_beamsolver){

    double real, imag;
    double ztemp1=-2./dev_beamsolver.xks;
    double btper0;
    double btpar0;
    hipDoubleComplex ctmp = make_hipDoubleComplex(0, 0);
    for (unsigned int i=0; i < dev_beamsolver.numel;i++){
        sincos(rharm_arr[i] * theta, &real, &imag);
        cuCAddAss(ctmp, hipCmul(make_hipDoubleComplex(real, -imag), rpart[i]));
    }
    btper0 = btpar+ztemp1*ctmp.x;
    btpar0 = sqrt(1.-btper0/(gamma*gamma));
    k2pp+= dev_beamsolver.xks*(1.-1./btpar0)+dev_beamsolver.xku;
    k2gg+= ctmp.y/btpar0/gamma;
}

__device__ void dev_rungekutta(double& gamma, double& theta, double btpar, hipDoubleComplex* rpart, DeviceBeamSolver dev_beamsolver) {

    double k2gg=0;
    double k2pp=0;
    double k3gg;
    double k3pp;
    double stpz=0.5 * dev_beamsolver.delz;
    dev_run_ODE(gamma,theta, k2pp, k2gg, btpar, rpart, dev_beamsolver);
    gamma+=stpz*k2gg;
    theta+=stpz*k2pp;
    k3gg=k2gg;
    k3pp=k2pp;
    k2gg=0;
    k2pp=0;
    dev_run_ODE(gamma,theta, k2pp, k2gg, btpar, rpart, dev_beamsolver);
    gamma+=stpz*(k2gg-k3gg);
    theta+=stpz*(k2pp-k3pp);
    k3gg/=6;
    k3pp/=6;
    k2gg*=-0.5;
    k2pp*=-0.5;
    dev_run_ODE(gamma,theta, k2pp, k2gg, btpar, rpart, dev_beamsolver);
    stpz=dev_beamsolver.delz;
    gamma+=stpz*k2gg;
    theta+=stpz*k2pp;
    k3gg-=k2gg;
    k3pp-=k2pp;
    k2gg*=2;
    k2pp*=2;
    dev_run_ODE(gamma, theta, k2pp, k2gg, btpar, rpart, dev_beamsolver);
    gamma+=stpz*(k3gg+k2gg/6.0);
    theta+=stpz*(k3pp+k2pp/6.0);
}


