#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include <Beam.h>
#include <Field.h>
#include <Undulator.h>
#include <vector>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace std;

#define BLOCK_DIM 32

//__device__ __constant__ int nfld_arr[]; //1D int texture obj
//__device__ __constant__ double rtmp_arr[]; //1D double texture obj
//__device__ __constant__ double rharm_arr[];//1D double texture obj
//__device__ __constant__ int harm_arr[];   // 1D double texture obj
//__device__ __constant__ int ngrid_arr[];  // 1D int texture obj
//__device__ __constant__ int first_arr[];  // 1D int texture obj
//__device__ __constant__ double dgrid_arr[];  // 1D double texture obj
//__device__ __constant__ double gridmax_arr[];// 1D double texture obj

surface<void, cudaSurfaceType2DLayered> field_surf; // 3D double2(complex) texture obj


BeamSolver::BeamSolver()
{
  onlyFundamental=false;
}

BeamSolver::~BeamSolver(){}


void BeamSolver::advance(double delz, Beam* beam, vector<Field*>* field, Undulator* und) {

    bool onlyFundamental = false; //!!!! change, stub
    vector<int> nfld;
    vector<double> rtmp;
    vector<double> rharm;
    int xks=1;  // default value in the case that no field is defined
    int xku=und->getku();

    int* harm_host;   // 1D double texture obj
    int* ngrid_host;  // 1D int texture obj
    int* first_host;  // 1D int texture obj
    double* dgrid_host;  // 1D double texture obj
    double* gridmax_host;// 1D double texture obj

    hipChannelFormatDesc field_channel_desc = hipCreateChannelDesc<hipDoubleComplex>();
    hipArray* field_array_ref;
    hipDoubleComplex* field_array_data_host;
    hipMemcpy3DParms field_memcpy_parms = {0};
    vector<SolverComputation> solver_computations;
    
    if (xku==0){   // in the case of drifts - the beam stays in phase if it has the reference energy // this requires that the phase slippage is not applied
        xku=xks*0.5/und->getGammaRef()/und->getGammaRef();
    }

    for (int i=0; i < field->size(); i++){
        int harm=field->at(i)->getHarm();
        if ((harm==1) || !onlyFundamental){
            xks=field->at(i)->xks/static_cast<double>(harm);    // fundamental field wavenumber used in ODE below
            nfld.push_back(i);
            rtmp.push_back(und->fc(harm)/field->at(i)->xks);      // here the harmonics have to be taken care
            rharm.push_back(static_cast<double>(harm));
        }
    }  

    double aw=und->getaw();
    double autophase=und->autophase();

    DeviceBeamSolver dev_beamsolver;
    dev_beamsolver.delz = delz;
    dev_beamsolver.xks = xks;
    dev_beamsolver.xku = xku;
    dev_beamsolver.numel = field->size();

    DeviceUndulator dev_undulator;
    dev_undulator.aw = und->aw[und->istepz];  
    dev_undulator.ax = und->ax[und->istepz];               
    dev_undulator.ay = und->ay[und->istepz];
    dev_undulator.kx = und->kx[und->istepz];
    dev_undulator.ky = und->ky[und->istepz];
    dev_undulator.gradx = und->gradx[und->istepz]; 
    dev_undulator.grady = und->grady[und->istepz];
    dev_undulator.autophase = autophase;

    DeviceField dev_field;
    dev_field.numel = field->size();
    dev_field.xsize = field->at(0)->field.size();
    dev_field.ysize = field->at(0)->field.at(0).size();

    gpuErrchk(hipMalloc(&nfld_arr, nfld.size()*sizeof(int)));	
    gpuErrchk(hipMalloc(&rtmp_arr, rtmp.size()*sizeof(double)));	
    gpuErrchk(hipMalloc(&rharm_arr, rharm.size()*sizeof(double)));	
    
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(nfld_arr), nfld.data(), sizeof(int)*nfld.size()));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(rtmp_arr), rtmp.data(), sizeof(double)*rtmp.size()));
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(rharm_arr), rharm.data(), sizeof(double)*rharm.size()));

    gpuErrchk(hipHostMalloc(&harm_host, field->size()*sizeof(int)));	
    gpuErrchk(hipHostMalloc(&ngrid_host, field->size()*sizeof(int)));	
    gpuErrchk(hipHostMalloc(&first_host, field->size()*sizeof(int)));	
    gpuErrchk(hipHostMalloc(&dgrid_host, field->size()*sizeof(double)));	
    gpuErrchk(hipHostMalloc(&gridmax_host, field->size()*sizeof(double)));	
    
    field_array_data_host = (hipDoubleComplex*)malloc(sizeof(hipDoubleComplex)*dev_field.xsize*dev_field.ysize*dev_field.numel);

    for (int i = 0; i < field->size(); ++i) {
        Field* field_el = field->at(i);
        harm_host[i] = field_el->harm; 
        ngrid_host[i] = field_el->ngrid;   
        first_host[i] = field_el->first;   
        dgrid_host[i] = field_el->dgrid;  
        gridmax_host[i] = field_el->gridmax;

        for (int j = 0;  j < dev_field.xsize; j++) {
            
            memcpy((void*)(field_array_data_host+dev_field.ysize*(j+dev_field.xsize*i)), field_el->field[j].data(), sizeof(hipDoubleComplex)*dev_field.ysize);            
        }
    }
    
    hipMalloc3DArray(&field_array_ref, &field_channel_desc, make_hipExtent(sizeof(hipDoubleComplex)*dev_field.xsize, dev_field.ysize, dev_field.numel), hipArrayLayered);
    field_memcpy_parms.srcPos = make_hipPos(0,0,0);
    field_memcpy_parms.dstPos = make_hipPos(0,0,0);
    field_memcpy_parms.srcPtr = make_hipPitchedPtr(field_array_data_host, dev_field.xsize * sizeof(hipDoubleComplex), dev_field.ysize, dev_field.numel);
    field_memcpy_parms.dstArray = field_array_ref;
    field_memcpy_parms.extent = make_hipExtent(dev_field.xsize, dev_field.ysize, dev_field.numel);
    field_memcpy_parms.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&field_memcpy_parms);
    cudaBindSurfaceToArray(field_surf, field_array_ref, field_channel_desc);

    hipMemcpyToSymbol(HIP_SYMBOL(harm_arr), harm_host, field->size()*sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(ngrid_arr), ngrid_host, field->size()*sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(first_arr), first_host, field->size()*sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(dgrid_arr), dgrid_host, field->size()*sizeof(double), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(gridmax_arr), gridmax_host, field->size()*sizeof(double), 0, hipMemcpyHostToDevice);

    for (unsigned int i = 0; i < beam->beam.size(); ++i) {
        unsigned int group_len = beam->beam[i].size();
        unsigned int n_blocks = group_len / BLOCK_DIM;
        SolverComputation computation;
        if ((group_len % BLOCK_DIM) != 0) n_blocks++;
        hipStreamCreate(&computation.stream);
        hipMalloc(&computation.part_group, n_blocks*BLOCK_DIM*sizeof(Particle));     
        hipMemcpyAsync(computation.part_group, beam->beam[i].data(), sizeof(Particle)*group_len, hipMemcpyHostToDevice, computation.stream);
        dev_advance<<<n_blocks, BLOCK_DIM, 0, computation.stream>>>(dev_beamsolver, dev_undulator, dev_field, computation.part_group, group_len);
        hipMemcpyAsync(beam->beam[i].data(), computation.part_group, sizeof(Particle)*group_len, hipMemcpyDeviceToHost, computation.stream);
        solver_computations.push_back(computation);
    }

    for (unsigned int i = 0; i < beam->beam.size(); ++i) {
        SolverComputation computation = solver_computations[i];
        hipStreamSynchronize(computation.stream);
        hipFree(computation.part_group);
    }
    // No surface unbind procedures. MEH!
    hipFreeArray(field_array_ref);
    free(field_array_data_host);
    hipHostFree(harm_host);
    hipHostFree(ngrid_host);
    hipHostFree(first_host);
    hipHostFree(dgrid_host);
    hipHostFree(gridmax_host);
}
